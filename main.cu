#include "hip/hip_runtime.h"
/* main.cu */
#include <iostream>
#include "renderer/renderer.h"
#include "processor/processor.h"

__global__
void swap(int& px, int& py)
{
    int tmp = px;
    px = py;
    py = tmp;
}

int main()
{
    int* x;
    int* y;

    hipMallocManaged(&x, sizeof(int));
    hipMallocManaged(&y, sizeof(int));

    *x = 5;
    *y = 8;

    std::cout << "Hello" << std::endl;
    std::cout << "GPU process... : " << *x << "  " << *y << std::endl;

    swap<<<1, 1>>>(*x, *y);
    pTest<<<1, 1>>>();
    hipDeviceSynchronize();

    test();

    std::cout << "Process ended -> x = " << *x << ", y = " << *y << std::endl;

    hipFree(x);
    hipFree(y);

    return 0;
}